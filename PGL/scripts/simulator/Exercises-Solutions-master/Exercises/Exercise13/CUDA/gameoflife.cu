//------------------------------------------------------------------------------
//
// Name:       gameoflife.cu
// 
// Purpose:    CUDA implementation of Conway's game of life
//
// HISTORY:    Written by Tom Deakin and Simon McIntosh-Smith, August 2013
//
//------------------------------------------------------------------------------

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define FINALSTATEFILE "final_state.dat"

// Define the state of the cell
#define DEAD  0
#define ALIVE 1

/*************************************************************************************
 * Forward declarations of utility functions
 ************************************************************************************/
void die(const char* message, const int line, const char *file);
void load_board(char* board, const char* file, const unsigned int nx, const unsigned int ny);
void print_board(const char* board, const unsigned int nx, const unsigned int ny);
void save_board(const char* board, const unsigned int nx, const unsigned int ny);
void load_params(const char *file, unsigned int *nx, unsigned int *ny, unsigned int *iterations);
void errorCheck(hipError_t error);

/*************************************************************************************
 * Game of Life worker method - CUDA kernel
 ************************************************************************************/

// Apply the rules of life to tick and save in tock
__global__ void accelerate_life(const char* tick, char* tock, const int nx, const int ny)
{
    // The cell we work on in the loop
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int idy = blockDim.y * blockIdx.y + threadIdx.y;

    // Index with respect to global array
    unsigned int id = idy * nx + idx;
    unsigned int id_b = (threadIdx.y + 1) * (blockDim.x + 2) + threadIdx.x + 1;

    // Copy block to shared memory
    extern __shared__ char block[];
    block[id_b] = tick[id];

    // Copy the halo cells (those around the block) to shared memory
    const unsigned int block_r = (blockIdx.x + 1) % gridDim.x;
    const unsigned int block_l = (blockIdx.x == 0) ? gridDim.x - 1 : blockIdx.x - 1;
    const unsigned int block_u = (blockIdx.y + 1) % gridDim.y;
    const unsigned int block_d = (blockIdx.y  == 0) ? gridDim.y - 1: blockIdx.y - 1;

    // Select the first row of threads
    if (threadIdx.y == 0)
    {
        // Down row
        block[threadIdx.x + 1] = tick[(blockDim.y * block_d + blockDim.y - 1) * nx + idx];
    }
    // Select the last row of threads
    if (threadIdx.y == blockDim.y - 1)
    {
        // Up row
        block[id_b + blockDim.x + 2] = tick[(blockDim.y * block_u) * nx + idx];
    }

    // Select right column of threads
    if (threadIdx.x == blockDim.x - 1)
    {
        // Copy in right
        block[id_b + 1] = tick[nx * idy + (blockDim.x * block_r)];
    }

    // Select left column of threads
    if (threadIdx.x == 0)
    {
        // Copy in left
        block[id_b - 1] = tick[nx * idy + (blockDim.x * block_l + blockDim.x - 1)];
    }


    // Add the 4 corner halo cells
    block[0] = tick[nx * (blockDim.y * block_d + blockDim.y - 1) + (blockDim.x * block_l) + blockDim.x - 1];
    block[blockDim.x + 1] = tick[nx * (blockDim.y * block_d + blockDim.y - 1) + (blockDim.x * block_r)];
    block[(blockDim.x + 2) * (blockDim.y + 1)] = tick[nx * (blockDim.y * block_u) + (blockDim.x * block_l) + blockDim.x - 1];
    block[(blockDim.x + 2) * (blockDim.y + 2) - 1] = tick[nx * (blockDim.y * block_u) + (blockDim.x * block_r)];
    
    __syncthreads();

    // Indexes of rows/columns next to id_b
    unsigned int x_l, x_r, y_u, y_d;

    // Calculate indexes
    x_r = threadIdx.x + 2;
    x_l = threadIdx.x;
    y_u = threadIdx.y + 2;
    y_d = threadIdx.y;

    // Count alive neighbours (out of eight)
    int neighbours = 0;
    if (block[(threadIdx.y + 1) * (blockDim.x + 2) + x_l] == ALIVE) neighbours++;
    if (block[y_u * (blockDim.x + 2) + x_l] == ALIVE) neighbours++;
    if (block[y_d * (blockDim.x + 2) + x_l] == ALIVE) neighbours++;
        
    if (block[(threadIdx.y + 1) * (blockDim.x + 2) + x_r] == ALIVE) neighbours++;
    if (block[y_u * (blockDim.x + 2) + x_r] == ALIVE) neighbours++;
    if (block[y_d * (blockDim.x + 2) + x_r] == ALIVE) neighbours++;
         
    if (block[y_u * (blockDim.x + 2) + threadIdx.x + 1] == ALIVE) neighbours++;
    if (block[y_d * (blockDim.x + 2) + threadIdx.x + 1] == ALIVE) neighbours++;

    // Apply game of life rules
    if (block[id_b] == ALIVE)
    {
        if (neighbours == 2 || neighbours == 3)
            // Cell lives on
            tock[id] = ALIVE;
        else
            // Cell dies by over/under population
            tock[id] = DEAD;
    }
    else
    {
        if (neighbours == 3)
            // Cell becomes alive through reproduction
            tock[id] = ALIVE;
        else
            // Remains dead
            tock[id] = DEAD;
    }

}


/*************************************************************************************
 * Main function
 ************************************************************************************/

int main(int argc, char **argv)
{

    // Check we have a starting state file
    if (argc != 5)
    {
        printf("Usage:\n./gameoflife input.dat input.params bx by\n");
        printf("\tinput.dat\tpattern file\n");
        printf("\tinput.params\tparameter file defining board size\n");
        printf("\tbx by\tsizes of thread blocks - must divide the board size equally\n");
        return EXIT_FAILURE;
    }


    // Board dimensions and iteration total
    unsigned int nx, ny;
    unsigned int iterations;
    unsigned int bx = atoi(argv[3]);
    unsigned int by = atoi(argv[4]);

    load_params(argv[2], &nx, &ny, &iterations);

    // Allocate memory for boards
    size_t size = nx * ny * sizeof(char);
    char* h_board = (char *)calloc(nx * ny, sizeof(char));
    char* d_board_tick;
    char* d_board_tock;

    errorCheck(hipMalloc(&d_board_tick, size));
    errorCheck(hipMalloc(&d_board_tock, size));

    // Load in the starting state to board_tick
    load_board(h_board, argv[1], nx, ny);

    // Display the starting state
    printf("Starting state\n");
    print_board(h_board, nx, ny);

    // Copy the host array to the device array
    errorCheck(hipMemcpy(d_board_tick, h_board, size, hipMemcpyHostToDevice));

    // Define our problem size for CUDA
    dim3 numBlocks(nx/bx, ny/by);
    dim3 numThreads(bx, by);
    size_t sharedMem = sizeof(char) * (bx + 2) * (by + 2);

    // Loop
    for (unsigned int i = 0; i < iterations; i++)
    {
        // Apply the rules of Life
        accelerate_life<<<numBlocks, numThreads, sharedMem>>>(d_board_tick, d_board_tock, nx, ny);
        errorCheck(hipPeekAtLastError());

        // Swap the boards over
        char *tmp = d_board_tick;
        d_board_tick = d_board_tock;
        d_board_tock = tmp;
    }

    // Copy the device array back to the host
    errorCheck(hipMemcpy(h_board, d_board_tick, size, hipMemcpyDeviceToHost));

    // Display the final state
    printf("Finishing state\n");
    print_board(h_board, nx, ny);

    // Save the final state of the board
    save_board(h_board, nx, ny);

    return EXIT_SUCCESS;
}


/*************************************************************************************
 * Utility functions
 ************************************************************************************/

// Function to load the params file and set up the X and Y dimensions
void load_params(const char* file, unsigned int *nx, unsigned int *ny, unsigned int *iterations)
{
    FILE *fp = fopen(file, "r");
    if (!fp)
        die("Could not open params file.", __LINE__, __FILE__);

    int retval;
    retval = fscanf(fp, "%d\n", nx);
    if (retval != 1)
        die("Could not read params file: nx.", __LINE__, __FILE__);
    retval = fscanf(fp, "%d\n", ny);
    if (retval != 1)
        die("Could not read params file: ny", __LINE__, __FILE__);
    retval = fscanf(fp, "%d\n", iterations);
    if (retval != 1)
        die("Could not read params file: iterations", __LINE__, __FILE__);

    fclose(fp);
}

// Function to load in a file which lists the alive cells
// Each line of the file is expected to be: x y 1
void load_board(char* board, const char* file, const unsigned int nx, const unsigned int ny)
{
    FILE *fp = fopen(file, "r");
    if (!fp)
        die("Could not open input file.", __LINE__, __FILE__);

    int retval;
    unsigned int x, y, s;
    while ((retval = fscanf(fp, "%d %d %d\n", &x, &y, &s)) != EOF)
    {
        if (retval != 3)
            die("Expected 3 values per line in input file.", __LINE__, __FILE__);
        if (x > nx - 1)
            die("Input x-coord out of range.", __LINE__, __FILE__);
        if (y > ny - 1)
            die("Input y-coord out of range.", __LINE__, __FILE__);
        if (s != ALIVE)
            die("Alive value should be 1.", __LINE__, __FILE__);

        board[x + y * nx] = ALIVE;
    }

    fclose(fp);
}

// Function to print out the board to stdout
// Alive cells are displayed as O
// Dead cells are displayed as .
void print_board(const char* board, const unsigned int nx, const unsigned int ny)
{
    for (unsigned int i = 0; i < ny; i++)
    {
        for (unsigned int j = 0; j < nx; j++)
        {
            if (board[i * nx + j] == DEAD)
                printf(".");
            else
                printf("O");
        }
        printf("\n");
    }
}

void save_board(const char* board, const unsigned int nx, const unsigned int ny)
{
    FILE *fp = fopen(FINALSTATEFILE, "w");
    if (!fp)
        die("Could not open final state file.", __LINE__, __FILE__);

    for (unsigned int i = 0; i < ny; i++)
    {
        for (unsigned int j = 0; j < nx; j++)
        {
            if (board[i * nx + j] == ALIVE)
                fprintf(fp, "%d %d %d\n", j, i, ALIVE);
        }
    }
}

void errorCheck(hipError_t error)
{
    if (error != hipSuccess)
        die(hipGetErrorString(error), __LINE__, __FILE__);
}

// Function to display error and exit nicely
void die(const char* message, const int line, const char *file)
{
  fprintf(stderr, "Error at line %d of file %s:\n", line, file);
  fprintf(stderr, "%s\n",message);
  fflush(stderr);
  exit(EXIT_FAILURE);
}
