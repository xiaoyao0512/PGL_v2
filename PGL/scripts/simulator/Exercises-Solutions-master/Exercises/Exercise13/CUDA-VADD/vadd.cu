//------------------------------------------------------------------------------
//
// Name:       vadd.cu
// 
// Purpose:    CUDA implementation of VADD
//
// HISTORY:    Written by Tom Deakin and Simon McIntosh-Smith, August 2013
//
//------------------------------------------------------------------------------

#include <stdio.h>
#include <hip/hip_runtime.h>

#define TOL    (0.001)   // tolerance used in floating point comparisons
#define LENGTH (1024)    // length of vectors a, b, and c

/*************************************************************************************
 * CUDA kernel
 ************************************************************************************/

__global__ void vadd(const float* a,
					 const float* b,
					       float* c,
					 const unsigned int count)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < count) {
		c[i] = a[i] + b[i];
	}
}

/*************************************************************************************
 * Main function
 ************************************************************************************/

int main(void)
{
    float        h_a[LENGTH];       // a vector
    float        h_b[LENGTH];       // b vector
    float        h_c[LENGTH];       // c vector (a+b) returned from the compute device
    float *d_a, *d_b, *d_c;         // CUDA memory
    unsigned int correct;           // number of correct results

    // Fill vectors a and b with random float values
    int i = 0;
    int count = LENGTH;
    for(i = 0; i < count; i++){
        h_a[i] = rand() / (float)RAND_MAX;
        h_b[i] = rand() / (float)RAND_MAX;
    }

    // Allocate CUDA memory
    hipMalloc(&d_a, sizeof(float) * LENGTH);
    hipMalloc(&d_b, sizeof(float) * LENGTH);
    hipMalloc(&d_c, sizeof(float) * LENGTH);

    // Write buffers a and b to GPU memory
    hipMemcpy(d_a, h_a, sizeof(float) * LENGTH, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float) * LENGTH, hipMemcpyHostToDevice);

    dim3 numBlocks(LENGTH);
    dim3 numThreads(1);
    vadd<<<numBlocks, numThreads>>>(d_a, d_b, d_c, LENGTH);

    // Copy result array back to host memory
    hipMemcpy(h_c, d_c, sizeof(float) * LENGTH, hipMemcpyDeviceToHost);

    // Test the results
    correct = 0;
    float tmp;
    
    for(i = 0; i < count; i++)
    {
        tmp = h_a[i] + h_b[i];     // assign element i of a+b to tmp
        tmp -= h_c[i];             // compute deviation of expected and output result
        if(tmp*tmp < TOL*TOL)        // correct if square deviation is less than tolerance squared
            correct++;
        else {
            printf(" tmp %f h_a %f h_b %f h_c %f \n",tmp, h_a[i], h_b[i], h_c[i]);
        }
    }
    
    // summarize results
    printf("C = A+B:  %d out of %d results were correct.\n", correct, count);

	return EXIT_SUCCESS;
}
